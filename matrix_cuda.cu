#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

typedef struct {
    int width;
    int height;
    float* elements;
} Matrix;


#define SIZE 10000
#define M_SIZE 100000000
#define BLOCK_SIZE 10

__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);

void MatMul(const Matrix A, const Matrix B, Matrix C)
{
    Matrix d_A;
    d_A.width = A.width; d_A.height = A.height;
    size_t size = A.width * A.height * sizeof(float);
    hipMalloc((void**) &d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size,
               hipMemcpyHostToDevice);
    Matrix d_B;
    d_B.width = B.width; d_B.height = B.height;
    size = B.width * B.height * sizeof(float);
    hipMalloc((void**) &d_B.elements, size);
    hipMemcpy(d_B.elements, B.elements, size,
               hipMemcpyHostToDevice);

    Matrix d_C;
    d_C.width = C.width; d_C.height = C.height;
    size = C.width * C.height * sizeof(float);
    hipMalloc((void**) &d_C.elements, size);

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

    hipMemcpy(C.elements, d_C.elements, size,
               hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
}

__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
    float Cvalue = 0;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    for (int e = 0; e < A.width; ++e)
        Cvalue += A.elements[row * A.width + e]
                * B.elements[e * B.width + col];
    C.elements[row * C.width + col] = Cvalue;
}

void gen_random_numbers(float *array, int len, int min, int max){
    for (int i = 0; i < len; i++)
        array[i] = rand() % (max - min + 1) + min;
}

int main() {
    Matrix A;
    A.width = SIZE;
    A.height = SIZE;
    float *A_numbers = (float*) malloc(sizeof(float)*M_SIZE);
    gen_random_numbers(A_numbers, M_SIZE, 10, 100);
    A.elements = A_numbers;

    Matrix B;
    B.width = SIZE;
    B.height = SIZE;
    float *B_numbers = (float*) malloc(sizeof(float)*M_SIZE);
    gen_random_numbers(B_numbers, M_SIZE, 10, 100);
    B.elements = B_numbers;

    Matrix C;
    C.width = SIZE;
    C.height = SIZE;
    float *C_numbers = (float*) malloc(sizeof(float)*M_SIZE);
    C.elements = C_numbers;

    
    clock_t start, finish;
    
    start = clock();
    MatMul(A, B, C);
    finish = clock();
    printf("Time = %f\n", ((float) (finish - start)) / CLOCKS_PER_SEC);

    return 0;
}
